#include "hip/hip_runtime.h"
#include "bincv-cuda/edge_filter.hpp"
#include <hip/hip_runtime.h>
#include <stdint.h>

namespace bincv {

__global__ void horizontalEdgeFilterKernel(const uint8_t* input, uint8_t* output, int width, int height, uint8_t threshold) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Skip out-of-bounds and border pixels
    if (x <= 0 || x >= width - 1 || y >= height)
        return;

    int idx = y * width + x;
    int left = input[y * width + (x - 1)];
    int right = input[y * width + (x + 1)];

    output[idx] = (abs(right - left) >= threshold) ? 1 : 0;
}

void runHorizontalEdgeFilter(const uint8_t* h_input, uint8_t* h_output, int width, int height, uint8_t threshold) {
    size_t imageSize = width * height * sizeof(uint8_t);
    uint8_t *d_input, *d_output;

    hipMalloc(&d_input, imageSize);
    hipMalloc(&d_output, imageSize);

    hipMemcpy(d_input, h_input, imageSize, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x,
                 (height + blockDim.y - 1) / blockDim.y);

    horizontalEdgeFilterKernel<<<gridDim, blockDim>>>(d_input, d_output, width, height, threshold);

    hipMemcpy(h_output, d_output, imageSize, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
}

}
